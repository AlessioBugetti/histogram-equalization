
#include <hip/hip_runtime.h>
/*
* SPDX-License-Identifier: GPL-3.0-only
 *
 * Author: Alessio Bugetti <alessiobugetti98@gmail.com>
 */

#define NUM_BINS 256

__global__ void
CalculateHistogram(const unsigned char* input,
                   unsigned int* histogram,
                   const unsigned int pixelCount)
{
    __shared__ unsigned int cache[NUM_BINS];
    if (threadIdx.x < NUM_BINS)
    {
        cache[threadIdx.x] = 0;
    }
    __syncthreads();

    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    while (tid < pixelCount)
    {
        atomicAdd(&(cache[input[tid]]), 1);
        tid += stride;
    }
    __syncthreads();

    if (threadIdx.x < NUM_BINS)
    {
        atomicAdd(&(histogram[threadIdx.x]), cache[threadIdx.x]);
    }
}

__global__ void
KoggeStoneScan(unsigned int* cdf, const unsigned int* histogram)
{
    __shared__ unsigned int cache[NUM_BINS];
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NUM_BINS)
    {
        cache[threadIdx.x] = histogram[tid];
    }
    else
    {
        cache[threadIdx.x] = 0;
    }

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        __syncthreads();
        unsigned int temp = cache[threadIdx.x];
        if (threadIdx.x >= stride)
        {
            temp += cache[threadIdx.x - stride];
        }
        __syncthreads();

        if (threadIdx.x >= stride)
        {
            cache[threadIdx.x] = temp;
        }
    }

    if (tid < NUM_BINS)
    {
        cdf[tid] = cache[threadIdx.x];
    }
}

__global__ void
KoggeStoneScanDoubleBuffer(unsigned int* cdf, const unsigned int* histogram)
{
    __shared__ unsigned int cache[NUM_BINS];
    __shared__ unsigned int cacheAux[NUM_BINS];

    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < NUM_BINS)
    {
        cache[threadIdx.x] = histogram[tid];
        cacheAux[threadIdx.x] = histogram[tid];
    }
    else
    {
        cache[threadIdx.x] = 0;
        cacheAux[threadIdx.x] = 0;
    }

    unsigned int* inputBuffer = cache;
    unsigned int* outputBuffer = cacheAux;

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        __syncthreads();

        unsigned int* temp = inputBuffer;
        inputBuffer = outputBuffer;
        outputBuffer = temp;

        if (threadIdx.x >= stride)
        {
            outputBuffer[threadIdx.x] =
                inputBuffer[threadIdx.x] + inputBuffer[threadIdx.x - stride];
        }
        else
        {
            outputBuffer[threadIdx.x] = inputBuffer[threadIdx.x];
        }
    }

    if (tid < NUM_BINS)
    {
        cdf[tid] = outputBuffer[threadIdx.x];
    }
}

__global__ void
BrentKungScan(unsigned int* cdf, const unsigned int* histogram)
{
    __shared__ unsigned int cache[NUM_BINS];
    unsigned int tid = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_BINS)
    {
        cache[threadIdx.x] = histogram[tid];
    }

    if (tid + blockDim.x < NUM_BINS)
    {
        cache[threadIdx.x + blockDim.x] = histogram[tid + blockDim.x];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
    {
        __syncthreads();

        unsigned int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < NUM_BINS)
        {
            cache[index] += cache[index - stride];
        }
    }

    for (unsigned int stride = NUM_BINS / 4; stride > 0; stride /= 2)
    {
        __syncthreads();

        unsigned int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < NUM_BINS)
        {
            cache[index + stride] += cache[index];
        }
    }

    __syncthreads();
    if (tid < NUM_BINS)
    {
        cdf[tid] = cache[threadIdx.x];
    }
    if (tid + blockDim.x < NUM_BINS)
    {
        cdf[tid + blockDim.x] = cache[threadIdx.x + blockDim.x];
    }
}

__global__ void
NormalizeCdf(unsigned int* cdf, const unsigned int cdfMin, const unsigned int pixelCount)
{
    if (const unsigned int tid = threadIdx.x; tid < NUM_BINS)
    {
        cdf[tid] = __double2int_rn(static_cast<double>(cdf[tid] - cdfMin) / (pixelCount - cdfMin) *
                                   (NUM_BINS - 1));
    }
}

__global__ void
EqualizeHistogram(unsigned char* output,
                  const unsigned char* input,
                  const unsigned int* cdf,
                  const unsigned int pixelCount)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    while (tid < pixelCount)
    {
        output[tid] = cdf[input[tid]];
        tid += stride;
    }
}